#include "hip/hip_runtime.h"
# include <cmath>
# include <iostream>

# include <hip/hip_runtime.h>
# include <hipblas.h>
# include <hipsparse.h>

# include "cgls.cuh"

void sparse_cgls_gpu(int * iA, int * jA, float * vA, float * B, float * x, int N, int M, int NNZ, int NIT, float TOL)
{
    /* Get handle to the CUBLAS context */
    hipblasHandle_t cublasHandle = 0;
    hipblasStatus_t hipblasStatus_t;
    hipblasStatus_t = hipblasCreate(&cublasHandle);
    if (hipblasStatus_t != HIPBLAS_STATUS_SUCCESS) printf("Error cublas\n");

    /* Get handle to the CUSPARSE context */
    hipsparseHandle_t cusparseHandle = 0;
    hipsparseStatus_t cusparseStatus;
    cusparseStatus = hipsparseCreate(&cusparseHandle);
    if (cusparseStatus != HIPSPARSE_STATUS_SUCCESS) printf("Error cusparse\n");

    size_t bsize;
    void * buffer;
    float beta = 0.0f;
    float alpha = 1.0f;
    float a, b, qTq, rTr, rd;

    int * d_iA_coo; hipMalloc((void **)&d_iA_coo, NNZ * sizeof(int));
    int * d_iA_csr; hipMalloc((void **)&d_iA_csr,(N+1)* sizeof(int));

    hipMemcpy(d_iA_coo, iA, NNZ * sizeof(int), hipMemcpyHostToDevice);

    hipsparseXcoo2csr(cusparseHandle, d_iA_coo, NNZ, N, d_iA_csr, HIPSPARSE_INDEX_BASE_ZERO);

    hipFree(d_iA_coo);
		
    float * d_p; hipMalloc((void **)&d_p, M * sizeof(float)); 
    float * d_q; hipMalloc((void **)&d_q, N * sizeof(float));  
    float * d_r; hipMalloc((void **)&d_r, M * sizeof(float)); 
    float * d_s; hipMalloc((void **)&d_s, N * sizeof(float)); 
    float * d_x; hipMalloc((void **)&d_x, M * sizeof(float)); 

    float * d_vA; hipMalloc((void **)&d_vA, NNZ * sizeof(float));     
    int * d_jA_coo; hipMalloc((void **)&d_jA_coo, NNZ * sizeof(int)); 

    hipMemset(d_x, 0, M * sizeof(float));    
    hipMemset(d_p, 0, M * sizeof(float));
    hipMemset(d_q, 0, N * sizeof(float));
    hipMemset(d_r, 0, M * sizeof(float));
    hipMemcpy(d_s, B, N * sizeof(float), hipMemcpyHostToDevice);

    hipMemcpy(d_vA, vA, NNZ * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_jA_coo, jA, NNZ * sizeof(int), hipMemcpyHostToDevice);

    hipsparseDnVecDescr_t Dn_p;    
    hipsparseDnVecDescr_t Dn_q;    
    hipsparseDnVecDescr_t Dn_r;    
    hipsparseDnVecDescr_t Dn_s;    
    hipsparseSpMatDescr_t Sp_matA; 

    hipsparseCreateCsr(&Sp_matA, N, M, NNZ, d_iA_csr, d_jA_coo, d_vA, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F);
    hipsparseCreateDnVec(&Dn_p, M, d_p, HIP_R_32F);
    hipsparseCreateDnVec(&Dn_q, N, d_q, HIP_R_32F);
    hipsparseCreateDnVec(&Dn_r, M, d_r, HIP_R_32F);
    hipsparseCreateDnVec(&Dn_s, N, d_s, HIP_R_32F);

    hipsparseSpMV_bufferSize(cusparseHandle, HIPSPARSE_OPERATION_TRANSPOSE, &alpha, Sp_matA, Dn_s, &beta, Dn_r, HIP_R_32F, HIPSPARSE_SPMV_CSR_ALG1, &bsize);
    hipMalloc(&buffer, bsize);

    hipsparseSpMV(cusparseHandle, HIPSPARSE_OPERATION_TRANSPOSE, &alpha, Sp_matA, Dn_s, &beta, Dn_r, HIP_R_32F, HIPSPARSE_SPMV_CSR_ALG1, buffer);
    hipDeviceSynchronize();    

    hipblasScopy(cublasHandle, M, d_r, 1, d_p, 1);

    hipsparseSpMV(cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, Sp_matA, Dn_p, &beta, Dn_q, HIP_R_32F, HIPSPARSE_SPMV_CSR_ALG1, buffer);
    hipDeviceSynchronize();    

    for (int iteration = 0; iteration < NIT; iteration++)
    {
        qTq = 0.0f;
        hipblasSdot(cublasHandle, N, d_q, 1, d_q, 1, &qTq);
        hipDeviceSynchronize();    // qTq = q' * q

        rTr = 0.0f;
        hipblasSdot(cublasHandle, M, d_r, 1, d_r, 1, &rTr);
        hipDeviceSynchronize();    // rTr = r' * r 

        a = rTr / qTq;              // a = (r' * r) / (q' * q)
        hipblasSaxpy(cublasHandle, M, &a, d_p, 1, d_x, 1);
        hipDeviceSynchronize();    // x = x + a * p

        a *= -1.0f;
        hipblasSaxpy(cublasHandle, N, &a, d_q, 1, d_s, 1);
        hipDeviceSynchronize();    // s = s - a * q 

        rd = 0.0f;
        hipblasSdot(cublasHandle, M, d_r, 1, d_r, 1, &rd);
        hipDeviceSynchronize();    // rd = r' * r

        if (sqrtf(rd) < TOL) break; // Convergence condition 

        hipsparseSpMV(cusparseHandle, HIPSPARSE_OPERATION_TRANSPOSE, &alpha, Sp_matA, Dn_s, &beta, Dn_r, HIP_R_32F, HIPSPARSE_SPMV_CSR_ALG1, buffer);
        hipDeviceSynchronize();   // r = G' * s    

        rTr = 0.0f;
        hipblasSdot(cublasHandle, M, d_r, 1, d_r, 1, &rTr);
        hipDeviceSynchronize();   // rTr = r' * r 

        b = rTr / rd;              // b = (r' * r) / rd  
        hipblasSscal(cublasHandle, M, &b, d_p, 1);
        hipDeviceSynchronize();   // p = b * p  

        b = 1.0f;
        hipblasSaxpy(cublasHandle, M, &b, d_r, 1, d_p, 1);
        hipDeviceSynchronize();   // p += r  <---> p = r + b * p  
        
        hipsparseSpMV(cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, Sp_matA, Dn_p, &beta, Dn_q, HIP_R_32F, HIPSPARSE_SPMV_CSR_ALG1, buffer);
        hipDeviceSynchronize();   // q = G * p    
    }

    hipMemcpy(x, d_x, M * sizeof(float), hipMemcpyDeviceToHost);

    hipsparseDestroyDnVec(Dn_p);
    hipsparseDestroyDnVec(Dn_q);
    hipsparseDestroyDnVec(Dn_r);
    hipsparseDestroyDnVec(Dn_s);
    hipsparseDestroySpMat(Sp_matA);

    hipFree(d_vA);
    hipFree(d_iA_csr);
    hipFree(d_jA_coo);

    hipFree(d_x);
    hipFree(d_p);
    hipFree(d_q);
    hipFree(d_r);
    hipFree(d_s);

    hipsparseDestroy(cusparseHandle);
    hipblasDestroy(cublasHandle);
}


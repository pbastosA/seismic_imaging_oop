#include "hip/hip_runtime.h"
# include <cmath>
# include <iostream>

# include <hip/hip_runtime.h>
# include <hipblas.h>
# include <hipsparse.h>

# include "cgls.cuh"

void sparse_cgls_cpu(int * iA, int * jA, float * vA, float * B, float * x, int N, int M, int NNZ, int NIT, float TOL)
{
    float a, b, qTq, rTr, rd;

    float * s = new float[N]();
    float * q = new float[N]();
    float * r = new float[M]();
    float * p = new float[M]();

    // s = d - A * x
    for (int row = 0; row < N; row++) 
        s[row] = B[row]; 

    // r = A' * s    
    for (int ind = 0; ind < NNZ; ind++) 
        r[jA[ind]] += vA[ind] * s[iA[ind]];        

    // p = r and x = 0
    for (int col = 0; col < M; col++) 
    {
        x[col] = 0.0f;
        p[col] = r[col]; 
    }

    // q = A * p
    for (int ind = 0; ind < NNZ; ind++) 
        q[iA[ind]] += vA[ind] * p[jA[ind]];        

    // Iterations loop
    for (int iteration = 0; iteration < NIT; iteration++)
    {
        qTq = 0.0f;
        for (int row = 0; row < N; row++)             // q inner product
            qTq += q[row] * q[row];                   // qTq = q' * q

        rTr = 0.0f;
        for (int col = 0; col < M; col++)             // r inner product
            rTr += r[col] * r[col];                   // rTr = r' * r 

        a = rTr / qTq;                                // a = (r' * r) / (q' * q)                    

        for (int col = 0; col < M; col++)             // model atualization
            x[col] += a * p[col];                     // x = x + a * p

        for (int row = 0; row < N; row++)             // s atualization  
            s[row] -= a * q[row];                     // s = s - a * q 

        rd = 0.0f;
        for (int col = 0; col < M; col++)             // r inner product for division 
            rd += r[col] * r[col];                    // rd = r' * r

        if (sqrtf(rd) < TOL) break;                   // Convergence condition

        for (int col = 0; col < M; col++)             // Zeroing r 
            r[col] = 0.0f;                            // r = 0, for multiplication

        for (int index = 0; index < NNZ; index++)     // r atualization 
            r[jA[index]] += vA[index] * s[iA[index]]; // r = G' * s    
                
        rTr = 0.0f;    
        for (int col = 0; col < M; col++)             // r inner product
            rTr += r[col] * r[col];                   // rTr = r' * r
        
        b = rTr / rd;                                 // b = (r' * r) / rd

        for (int col = 0; col < M; col++)
            p[col] = r[col] + b * p[col];             // p = r + b * p 

        for (int row = 0; row < N; row++) 
            q[row] = 0.0f;                            // q = 0, for multiplication

        for (int index = 0; index < NNZ; index++) 
            q[iA[index]] += vA[index] * p[jA[index]]; // q = G * p           
    }

    delete[] s; delete[] q; delete[] r; delete[] p;
}

void sparse_cgls_gpu(int * iA, int * jA, float * vA, float * B, float * x, int N, int M, int NNZ, int NIT, float TOL)
{
    /* Get handle to the CUBLAS context */
    hipblasHandle_t cublasHandle = 0;
    hipblasStatus_t hipblasStatus_t;
    hipblasStatus_t = hipblasCreate(&cublasHandle);
    if (hipblasStatus_t != HIPBLAS_STATUS_SUCCESS) printf("Error cublas\n");

    /* Get handle to the CUSPARSE context */
    hipsparseHandle_t cusparseHandle = 0;
    hipsparseStatus_t cusparseStatus;
    cusparseStatus = hipsparseCreate(&cusparseHandle);
    if (cusparseStatus != HIPSPARSE_STATUS_SUCCESS) printf("Error cusparse\n");

    size_t bsize;
    void * buffer;
    float beta = 0.0f;
    float alpha = 1.0f;
    float a, b, qTq, rTr, rd;

    int * d_iA_coo; hipMalloc((void **)&d_iA_coo, NNZ * sizeof(int));
    int * d_iA_csr; hipMalloc((void **)&d_iA_csr,(N+1)* sizeof(int));

    hipMemcpy(d_iA_coo, iA, NNZ * sizeof(int), hipMemcpyHostToDevice);

    hipsparseXcoo2csr(cusparseHandle, d_iA_coo, NNZ, N, d_iA_csr, HIPSPARSE_INDEX_BASE_ZERO);

    hipFree(d_iA_coo);
		
    float * d_p; hipMalloc((void **)&d_p, M * sizeof(float)); 
    float * d_q; hipMalloc((void **)&d_q, N * sizeof(float));  
    float * d_r; hipMalloc((void **)&d_r, M * sizeof(float)); 
    float * d_s; hipMalloc((void **)&d_s, N * sizeof(float)); 
    float * d_x; hipMalloc((void **)&d_x, M * sizeof(float)); 

    float * d_vA; hipMalloc((void **)&d_vA, NNZ * sizeof(float));     
    int * d_jA_coo; hipMalloc((void **)&d_jA_coo, NNZ * sizeof(int)); 

    hipMemset(d_x, 0, M * sizeof(float));    
    hipMemset(d_p, 0, M * sizeof(float));
    hipMemset(d_q, 0, N * sizeof(float));
    hipMemset(d_r, 0, M * sizeof(float));
    hipMemcpy(d_s, B, N * sizeof(float), hipMemcpyHostToDevice);

    hipMemcpy(d_vA, vA, NNZ * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_jA_coo, jA, NNZ * sizeof(int), hipMemcpyHostToDevice);

    hipsparseDnVecDescr_t Dn_p;    
    hipsparseDnVecDescr_t Dn_q;    
    hipsparseDnVecDescr_t Dn_r;    
    hipsparseDnVecDescr_t Dn_s;    
    hipsparseSpMatDescr_t Sp_matA; 

    hipsparseCreateCsr(&Sp_matA, N, M, NNZ, d_iA_csr, d_jA_coo, d_vA, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F);
    hipsparseCreateDnVec(&Dn_p, M, d_p, HIP_R_32F);
    hipsparseCreateDnVec(&Dn_q, N, d_q, HIP_R_32F);
    hipsparseCreateDnVec(&Dn_r, M, d_r, HIP_R_32F);
    hipsparseCreateDnVec(&Dn_s, N, d_s, HIP_R_32F);

    hipsparseSpMV_bufferSize(cusparseHandle, HIPSPARSE_OPERATION_TRANSPOSE, &alpha, Sp_matA, Dn_s, &beta, Dn_r, HIP_R_32F, HIPSPARSE_SPMV_CSR_ALG1, &bsize);
    hipMalloc(&buffer, bsize);

    hipsparseSpMV(cusparseHandle, HIPSPARSE_OPERATION_TRANSPOSE, &alpha, Sp_matA, Dn_s, &beta, Dn_r, HIP_R_32F, HIPSPARSE_SPMV_CSR_ALG1, buffer);
    hipDeviceSynchronize();    

    hipblasScopy(cublasHandle, M, d_r, 1, d_p, 1);

    hipsparseSpMV(cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, Sp_matA, Dn_p, &beta, Dn_q, HIP_R_32F, HIPSPARSE_SPMV_CSR_ALG1, buffer);
    hipDeviceSynchronize();    

    for (int iteration = 0; iteration < NIT; iteration++)
    {
        qTq = 0.0f;
        hipblasSdot(cublasHandle, N, d_q, 1, d_q, 1, &qTq);
        hipDeviceSynchronize();    // qTq = q' * q

        rTr = 0.0f;
        hipblasSdot(cublasHandle, M, d_r, 1, d_r, 1, &rTr);
        hipDeviceSynchronize();    // rTr = r' * r 

        a = rTr / qTq;              // a = (r' * r) / (q' * q)
        hipblasSaxpy(cublasHandle, M, &a, d_p, 1, d_x, 1);
        hipDeviceSynchronize();    // x = x + a * p

        a *= -1.0f;
        hipblasSaxpy(cublasHandle, N, &a, d_q, 1, d_s, 1);
        hipDeviceSynchronize();    // s = s - a * q 

        rd = 0.0f;
        hipblasSdot(cublasHandle, M, d_r, 1, d_r, 1, &rd);
        hipDeviceSynchronize();    // rd = r' * r

        if (sqrtf(rd) < TOL) break; // Convergence condition 

        hipsparseSpMV(cusparseHandle, HIPSPARSE_OPERATION_TRANSPOSE, &alpha, Sp_matA, Dn_s, &beta, Dn_r, HIP_R_32F, HIPSPARSE_SPMV_CSR_ALG1, buffer);
        hipDeviceSynchronize();   // r = G' * s    

        rTr = 0.0f;
        hipblasSdot(cublasHandle, M, d_r, 1, d_r, 1, &rTr);
        hipDeviceSynchronize();   // rTr = r' * r 

        b = rTr / rd;              // b = (r' * r) / rd  
        hipblasSscal(cublasHandle, M, &b, d_p, 1);
        hipDeviceSynchronize();   // p = b * p  

        b = 1.0f;
        hipblasSaxpy(cublasHandle, M, &b, d_r, 1, d_p, 1);
        hipDeviceSynchronize();   // p += r  <---> p = r + b * p  
        
        hipsparseSpMV(cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, Sp_matA, Dn_p, &beta, Dn_q, HIP_R_32F, HIPSPARSE_SPMV_CSR_ALG1, buffer);
        hipDeviceSynchronize();   // q = G * p    
    }

    hipMemcpy(x, d_x, M * sizeof(float), hipMemcpyDeviceToHost);

    hipsparseDestroyDnVec(Dn_p);
    hipsparseDestroyDnVec(Dn_q);
    hipsparseDestroyDnVec(Dn_r);
    hipsparseDestroyDnVec(Dn_s);
    hipsparseDestroySpMat(Sp_matA);

    hipFree(d_vA);
    hipFree(d_iA_csr);
    hipFree(d_jA_coo);

    hipFree(d_x);
    hipFree(d_p);
    hipFree(d_q);
    hipFree(d_r);
    hipFree(d_s);

    hipsparseDestroy(cusparseHandle);
    hipblasDestroy(cublasHandle);
}


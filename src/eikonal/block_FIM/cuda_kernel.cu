#include "hip/hip_runtime.h"
# include <cmath> 
# include <vector>
# include <cassert>
# include <iostream>

# include "cuda_kernel.cuh"

void cuda_safe_call(hipError_t error)
{
    if(error != hipSuccess)
    {
        std::cout<<"CUDA error! "<<error<<"\n";
        exit(EXIT_FAILURE);
    }
}

void block_FIM_solver(CUDAMEMSTRUCT &cmem, bool verbose)
{
    int deviceID; 
    hipGetDevice(&deviceID);
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, deviceID);

	size_t freeMem, totalMem;
	hipMemGetInfo(&freeMem, &totalMem);

	std::cout << "Device id : "<<deviceID<<", name : "<<deviceProp.name<<"\n";	
    std::cout << "Total Memory : " << totalMem / (1024 * 1024) << "MB" << "\n";
    std::cout << "Free Memory  : " << freeMem / (1024 * 1024) << "MB" << "\n";

    int xdim = cmem.xdim;
    int ydim = cmem.ydim;
    int zdim = cmem.zdim;
    float dh = cmem.delta_h;    

    uint volsize = cmem.volsize;
    uint blknum = cmem.blknum;

    int nIter = cmem.nIter;
    uint nActiveBlock = cmem.nActiveBlock; // active list

    float * d_spd;
    float * d_sol;
    float * h_sol;
    float * t_sol;

    uint * d_list;
    bool * d_listVol;

    bool * d_con;
    bool * d_mask;

    // copy so that original value should not be modified
    uint *h_list = (uint*) malloc(blknum*sizeof(uint));
    bool *h_listed = (bool*) malloc(blknum*sizeof(bool));
    bool *h_listVol = (bool*) malloc(blknum*sizeof(bool));

    // initialization
    memcpy(h_list, cmem.h_list, blknum*sizeof(uint));
    memcpy(h_listed, cmem.h_listed, blknum*sizeof(bool));
    memcpy(h_listVol, cmem.h_listVol, blknum*sizeof(bool));

  	// create host/device memory using CUDA mem functions

	cuda_safe_call(hipMalloc((void**)&(d_spd), volsize*sizeof(float)));
	cuda_safe_call(hipMalloc((void**)&(d_sol), volsize*sizeof(float)));
	cuda_safe_call(hipMalloc((void**)&(t_sol), volsize*sizeof(float))); 
	cuda_safe_call(hipMalloc((void**)&(d_con), volsize*sizeof(bool)));  
	cuda_safe_call(hipMalloc((void**)&(d_list), blknum*sizeof(uint)));
	cuda_safe_call(hipMalloc((void**)&(d_listVol), blknum*sizeof(bool)));
	cuda_safe_call(hipMalloc((void**)&(d_mask), volsize*sizeof(bool)));

	cuda_safe_call(hipMemcpy(d_spd, cmem.h_spd, volsize*sizeof(float), hipMemcpyHostToDevice));
	cuda_safe_call(hipMemcpy(d_mask, cmem.h_mask, volsize*sizeof(bool), hipMemcpyHostToDevice));

    cuda_safe_call(hipMemcpy(d_list, h_list, nActiveBlock*sizeof(uint), hipMemcpyHostToDevice));
    cuda_safe_call(hipMemcpy(d_listVol, h_listVol, blknum*sizeof(bool), hipMemcpyHostToDevice));
    cuda_safe_call(hipMemcpy(d_sol, h_sol, volsize*sizeof(float), hipMemcpyHostToDevice));
    cuda_safe_call(hipMemcpy(t_sol, h_sol, volsize*sizeof(float), hipMemcpyHostToDevice));
    cuda_safe_call(hipMemset(d_con, 1, volsize*sizeof(bool)));

    // set dimension of block and entire grid size
    dim3 dimBlock(BLOCK_LENGTH,BLOCK_LENGTH,BLOCK_LENGTH);
    dim3 dimEntireGrid(blknum);
    dim3 dimGrid(nActiveBlock);

    int nTotalIter = 0;

    std::vector<int> sourceList;
    sourceList.push_back((zdim/2)*ydim*xdim + (ydim/2)*xdim + (xdim/2));

    uint nTotalBlockProcessed = 0;

    while(nActiveBlock > 0)
    {
        assert(nActiveBlock < 4294967295);

        nTotalBlockProcessed += nActiveBlock;

        nTotalIter++;

        // 1. run solver on current active tiles
        
        dimGrid.y = (unsigned int)floorf((float)(nActiveBlock-1)/65535)+1;
        dimGrid.x = (unsigned int)ceilf((float)nActiveBlock/(float)dimGrid.y);

        cuda_safe_call(hipMemcpy(d_list, h_list, nActiveBlock*sizeof(uint), hipMemcpyHostToDevice));
        
        run_solver<<<dimGrid,dimBlock>>>(d_spd, d_mask, d_sol, t_sol, d_con, d_list, xdim, ydim, zdim, dh, nIter, nActiveBlock);
        
        cuda_safe_call(hipGetLastError());       
        hipDeviceSynchronize();   

        // 2. reduction (only active tiles)

        run_reduction<<<dimGrid,dim3(BLOCK_LENGTH,BLOCK_LENGTH,BLOCK_LENGTH/2)>>>(d_con, d_listVol, d_list, nActiveBlock);
        
        cuda_safe_call(hipGetLastError());
        hipDeviceSynchronize();

        // 3. check neighbor tiles of converged tile
        // Add any active block of neighbor of converged block is inserted
        // to the list

        cuda_safe_call(hipMemcpy(h_listVol, d_listVol, blknum*sizeof(bool), hipMemcpyDeviceToHost));

        uint nBlkX = xdim/BLOCK_LENGTH;
        uint nBlkY = ydim/BLOCK_LENGTH;
        uint nOldActiveBlock = nActiveBlock;

        for(uint i = 0; i < nOldActiveBlock; i++)
        {
            // check 6-neighbor of current active tile
            uint currBlkIdx = h_list[i];

            if(!h_listVol[currBlkIdx]) // not active : converged
            {
                uint nb[6];
                nb[0] = (currBlkIdx < nBlkX*nBlkY) ? currBlkIdx : (currBlkIdx - nBlkX*nBlkY);  //tp
                nb[1] = ((currBlkIdx + nBlkX*nBlkY) >= blknum) ? currBlkIdx : (currBlkIdx + nBlkX*nBlkY); //bt
                nb[2] = (currBlkIdx < nBlkX) ? currBlkIdx : (currBlkIdx - nBlkX); //up
                nb[3] = ((currBlkIdx + nBlkX) >= blknum) ? currBlkIdx : (currBlkIdx + nBlkX); //dn
                nb[4] = (currBlkIdx%nBlkX == 0) ? currBlkIdx : currBlkIdx-1; //lf
                nb[5] = ((currBlkIdx+1)%nBlkX == 0) ? currBlkIdx : currBlkIdx+1; //rt

                for(int nbIdx = 0; nbIdx < 6; nbIdx++)
                {
                    uint currIdx = nb[nbIdx];

                    if(!h_listed[currIdx])
                    {
                        h_listed[currIdx] = true;
                        h_list[nActiveBlock++] = currIdx;
                    }
                }
            }
        }

        hipDeviceSynchronize();
    
        // 4. run solver only once for neighbor blocks of converged block
        // current active list contains active blocks and neighbor blocks of
        // any converged blocks.

        // update grid dimension because nActiveBlock is changed
        dimGrid.y = (unsigned int)floor(((float)nActiveBlock-1)/65535)+1;
        dimGrid.x = (unsigned int)ceil((float)nActiveBlock/(float)dimGrid.y);

        cuda_safe_call(hipMemcpy(d_list, h_list, nActiveBlock*sizeof(uint), hipMemcpyHostToDevice));
        
        run_check_neighbor<<< dimGrid, dimBlock >>>(d_spd, d_mask, t_sol, d_sol, d_con, d_list, xdim, ydim, zdim, dh, nOldActiveBlock, nActiveBlock);
        
        cuda_safe_call(hipGetLastError());
        hipDeviceSynchronize();

        // 5. reduction

        run_reduction<<<dimGrid,dim3(BLOCK_LENGTH,BLOCK_LENGTH,BLOCK_LENGTH/2)>>>(d_con, d_listVol, d_list, nActiveBlock);
        cuda_safe_call(hipGetLastError());
        hipDeviceSynchronize();

        // 6. update active list
        // read back active volume from the device and add
        // active block to active list on the host memory

        nActiveBlock = 0;
        cuda_safe_call(hipMemcpy(h_listVol, d_listVol, blknum*sizeof(bool), hipMemcpyDeviceToHost));

        for(uint i=0; i<blknum; i++)
        {
            if(h_listVol[i]) // true : active block (not converged)
            {
                h_listed[i] = true;
                h_list[nActiveBlock++] = i;
            }
            else
            { 
                h_listed[i] = false;
            }
        }

        hipDeviceSynchronize();
    }

	cuda_safe_call(hipMemcpy(cmem.h_sol, d_sol, volsize*sizeof(float), hipMemcpyDeviceToHost));

    free(h_list);
    free(h_listed);
    free(h_listVol);

    cuda_safe_call(hipFree(d_spd));
    cuda_safe_call(hipFree(d_sol));
    cuda_safe_call(hipFree(t_sol));  // temp solution for ping-pong
    cuda_safe_call(hipFree(d_con));  // convergence volume
    cuda_safe_call(hipFree(d_list));
    cuda_safe_call(hipFree(d_listVol));
    cuda_safe_call(hipFree(d_mask));
}

__device__ float get_time_eikonal(float a, float b, float c, float h, float s)
{
	float ret, tmp;

	// a > b > c
	if(a < b) { tmp = a; a = b; b = tmp; }
	if(b < c) { tmp = b; b = c; c = tmp; }
	if(a < b) { tmp = a; a = b; b = tmp; }

	ret = INF;

	if(c < INF)
	{
		ret = c + h*s;
		
		if(ret > b) 
		{	
			tmp = ((b+c) + sqrtf(2.0f*s*s*h*h - (b-c)*(b-c)))*0.5f;
		
			if(tmp > b) ret = tmp; 

			if(ret > a)	
			{				
                tmp = (a+b+c)/3.0f + sqrtf(2.0f*(a*(b-a) + b*(c-b) + c*(a-c)) + 3.0f*s*s*h*h) / 3.0f;

				if(tmp > a) ret = tmp;
			}
		}
	}

	return ret;
}

__global__ void run_solver(float* spd, bool* mask, const float *sol_in, float *sol_out, bool *con, uint* list, int xdim, int ydim, int zdim, float dh, int nIter, uint nActiveBlock)
{
	uint list_idx = blockIdx.y*gridDim.x + blockIdx.x;

	if(list_idx < nActiveBlock)
	{
		// retrieve actual block index from the active list
		uint block_idx = list[list_idx];

		float F;
		bool isValid;
		uint blocksize = BLOCK_LENGTH*BLOCK_LENGTH*BLOCK_LENGTH;
		uint base_addr = block_idx*blocksize;

		uint xgridlength = xdim/BLOCK_LENGTH;
		uint ygridlength = ydim/BLOCK_LENGTH;
		uint zgridlength = zdim/BLOCK_LENGTH;

		// compute block index
		uint bx = block_idx%xgridlength;
		uint tmpIdx = (block_idx - bx)/xgridlength;
		uint by = tmpIdx%ygridlength;
		uint bz = (tmpIdx-by)/ygridlength;

		uint tx = threadIdx.x;
		uint ty = threadIdx.y;
		uint tz = threadIdx.z;
		uint tIdx = tz*BLOCK_LENGTH*BLOCK_LENGTH + ty*BLOCK_LENGTH + tx;

		__shared__ float _sol[BLOCK_LENGTH+2][BLOCK_LENGTH+2][BLOCK_LENGTH+2];

		// copy global to shared memory
		dim3 idx(tx+1,ty+1,tz+1);

		SOL(idx.x,idx.y,idx.z) = sol_in[base_addr + tIdx];
		
        F = spd[base_addr + tIdx];
		
		isValid = mask[base_addr + tIdx];

		uint new_base_addr, new_tIdx;

		// 1-neighborhood values
		if(tx == 0) 
		{
			if(bx == 0) // end of the grid
			{	
				new_tIdx = tIdx;
				new_base_addr = base_addr;
			}
			else
			{
				new_tIdx = tIdx + BLOCK_LENGTH-1;
				new_base_addr = (block_idx - 1)*blocksize;	
			}

			SOL(tx,idx.y,idx.z) = sol_in[new_base_addr + new_tIdx];	
		}

		if(tx == BLOCK_LENGTH-1)
		{
			if(bx == xgridlength-1) // end of the grid
			{
				new_tIdx = tIdx;
				new_base_addr = base_addr;
			}
			else
			{
				new_tIdx = tIdx - (BLOCK_LENGTH-1);
				new_base_addr = (block_idx + 1)*blocksize;	
			}
			SOL(tx+2,idx.y,idx.z) = sol_in[new_base_addr + new_tIdx];	
		}

		if(ty == 0)
		{
			if(by == 0)
			{
				new_tIdx = tIdx;
				new_base_addr = base_addr;
			}
			else
			{
				new_tIdx = tIdx + (BLOCK_LENGTH-1)*BLOCK_LENGTH;
				new_base_addr = (block_idx - xgridlength)*blocksize;
			}

			SOL(idx.x,ty,idx.z) = sol_in[new_base_addr + new_tIdx];
		}

		if(ty == BLOCK_LENGTH-1)
		{
			if(by == ygridlength-1) 
			{
				new_tIdx = tIdx;
				new_base_addr = base_addr;
			}
			else
			{
				new_tIdx = tIdx - (BLOCK_LENGTH-1)*BLOCK_LENGTH;
				new_base_addr = (block_idx + xgridlength)*blocksize;
			}

			SOL(idx.x,ty+2,idx.z) = sol_in[new_base_addr + new_tIdx];
		}

		if(tz == 0)
		{
			if(bz == 0)
			{
				new_tIdx = tIdx;
				new_base_addr = base_addr;
			}
			else
			{
				new_tIdx = tIdx + (BLOCK_LENGTH-1)*BLOCK_LENGTH*BLOCK_LENGTH;
				new_base_addr = (block_idx - xgridlength*ygridlength)*blocksize;
			}

			SOL(idx.x,idx.y,tz) = sol_in[new_base_addr + new_tIdx];
		}

		if(tz == BLOCK_LENGTH-1)
		{
			if(bz == zgridlength-1) 
			{
				new_tIdx = tIdx;
				new_base_addr = base_addr;
			}
			else
			{
				new_tIdx = tIdx - (BLOCK_LENGTH-1)*BLOCK_LENGTH*BLOCK_LENGTH;
				new_base_addr = (block_idx + xgridlength*ygridlength)*blocksize;
			}

			SOL(idx.x,idx.y,tz+2) = sol_in[new_base_addr + new_tIdx];
		}

		__syncthreads();

		float a,b,c,oldT,newT;

		for(int iter=0; iter<nIter; iter++)	
		{
			// compute new value
			oldT = newT = SOL(idx.x,idx.y,idx.z);

			if(isValid)
			{
				a = min(SOL(tx,idx.y,idx.z),SOL(tx+2,idx.y,idx.z));
				b = min(SOL(idx.x,ty,idx.z),SOL(idx.x,ty+2,idx.z));
				c = min(SOL(idx.x,idx.y,tz),SOL(idx.x,idx.y,tz+2));

				float tmp = (float) get_time_eikonal(a, b, c, dh, F);

				newT = min(tmp,oldT);
			}
			__syncthreads();	

			if(isValid) SOL(idx.x,idx.y,idx.z) = newT;
		}

		float residue = oldT - newT;

		// write back to global memory
		con[base_addr + tIdx] = (residue < EPS) ? true : false;
		sol_out[base_addr + tIdx] = newT;		
	}
}

__global__ void run_reduction(bool *con, bool *listVol, uint *list, uint nActiveBlock)
{
	uint list_idx = blockIdx.y*gridDim.x + blockIdx.x;

	if(list_idx < nActiveBlock)
	{
		uint block_idx = list[list_idx];

		__shared__ bool conv[BLOCK_LENGTH*BLOCK_LENGTH*BLOCK_LENGTH];

		uint blocksize = BLOCK_LENGTH*BLOCK_LENGTH*BLOCK_LENGTH/2;
		uint base_addr = block_idx*blocksize*2;
		uint tx = threadIdx.x;
		uint ty = threadIdx.y;
		uint tz = threadIdx.z;
		uint tIdx = tz*BLOCK_LENGTH*BLOCK_LENGTH + ty*BLOCK_LENGTH + tx;

		conv[tIdx] = con[base_addr + tIdx];
		conv[tIdx + blocksize] = con[base_addr + tIdx + blocksize];

		__syncthreads();

		for(uint i=blocksize; i>0; i/=2)
		{
			if(tIdx < i)
			{
				bool b1, b2;
				b1 = conv[tIdx];
				b2 = conv[tIdx+i];
				conv[tIdx] = (b1 && b2) ? true : false ;
			}
			__syncthreads();
		}

        // active list is negation of tile convergence (active = not converged)

		if(tIdx == 0) listVol[block_idx] = !conv[0]; 
	}
}

__global__ void run_check_neighbor(float* spd, bool* mask, const float *sol_in, float *sol_out, bool *con, uint* list, int xdim, int ydim, int zdim, float dh, uint nActiveBlock, uint nTotalBlock)
{
	uint list_idx = blockIdx.y*gridDim.x + blockIdx.x;

	if(list_idx < nTotalBlock)
	{
		float F;
		bool isValid;
		
        __shared__ float _sol[BLOCK_LENGTH+2][BLOCK_LENGTH+2][BLOCK_LENGTH+2];

		uint block_idx = list[list_idx];
		uint blocksize = BLOCK_LENGTH*BLOCK_LENGTH*BLOCK_LENGTH;
		uint base_addr = block_idx*blocksize;

		uint tx = threadIdx.x;
		uint ty = threadIdx.y;
		uint tz = threadIdx.z;
		uint tIdx = tz*BLOCK_LENGTH*BLOCK_LENGTH + ty*BLOCK_LENGTH + tx;

		if(list_idx < nActiveBlock) // copy value
		{
			sol_out[base_addr + tIdx] = sol_in[base_addr + tIdx];
		} 
		else
		{
			uint xgridlength = xdim/BLOCK_LENGTH;
			uint ygridlength = ydim/BLOCK_LENGTH;
			uint zgridlength = zdim/BLOCK_LENGTH;

			// compute block index
			uint bx = block_idx%xgridlength;
			uint tmpIdx = (block_idx - bx)/xgridlength;
			uint by = tmpIdx%ygridlength;
			uint bz = (tmpIdx-by)/ygridlength;

			// copy global to shared memory
			dim3 idx(tx+1,ty+1,tz+1);
			
            _sol[idx.x][idx.y][idx.z] = sol_in[base_addr + tIdx];
			
            F = spd[base_addr + tIdx];
			
            if(F > 0) F = 1.0/F;
			
            isValid = mask[base_addr + tIdx];

			uint new_base_addr, new_tIdx;

			// 1-neighborhood values
			if(tx == 0) 
			{
				if(bx == 0) // end of the grid
				{	
					new_tIdx = tIdx;
					new_base_addr = base_addr;
				}
				else
				{
					new_tIdx = tIdx + BLOCK_LENGTH-1;
					new_base_addr = (block_idx - 1)*blocksize;	
				}
				_sol[tx][idx.y][idx.z] = sol_in[new_base_addr + new_tIdx];	
			}

			if(tx == BLOCK_LENGTH-1)
			{
				if(bx == xgridlength-1) // end of the grid
				{
					new_tIdx = tIdx;
					new_base_addr = base_addr;
				}
				else
				{
					new_tIdx = tIdx - (BLOCK_LENGTH-1);
					new_base_addr = (block_idx + 1)*blocksize;	
				}
				_sol[tx+2][idx.y][idx.z] = sol_in[new_base_addr + new_tIdx];	
			}

			if(ty == 0)
			{
				if(by == 0)
				{
					new_tIdx = tIdx;
					new_base_addr = base_addr;
				}
				else
				{
					new_tIdx = tIdx + (BLOCK_LENGTH-1)*BLOCK_LENGTH;
					new_base_addr = (block_idx - xgridlength)*blocksize;
				}
				_sol[idx.x][ty][idx.z] = sol_in[new_base_addr + new_tIdx];
			}

			if(ty == BLOCK_LENGTH-1) 
			{
				if(by == ygridlength-1) 
				{
					new_tIdx = tIdx;
					new_base_addr = base_addr;
				}
				else
				{
					new_tIdx = tIdx - (BLOCK_LENGTH-1)*BLOCK_LENGTH;
					new_base_addr = (block_idx + xgridlength)*blocksize;
				}
				_sol[idx.x][ty+2][idx.z] = sol_in[new_base_addr + new_tIdx];
			}

			if(tz == 0)
			{
				if(bz == 0)
				{
					new_tIdx = tIdx;
					new_base_addr = base_addr;
				}
				else
				{
					new_tIdx = tIdx + (BLOCK_LENGTH-1)*BLOCK_LENGTH*BLOCK_LENGTH;
					new_base_addr = (block_idx - xgridlength*ygridlength)*blocksize;
				}
				_sol[idx.x][idx.y][tz] = sol_in[new_base_addr + new_tIdx];
			}

			if(tz == BLOCK_LENGTH-1)
			{
				if(bz == zgridlength-1) // end of the grid
				{
					new_tIdx = tIdx;
					new_base_addr = base_addr;
				}
				else
				{
					new_tIdx = tIdx - (BLOCK_LENGTH-1)*BLOCK_LENGTH*BLOCK_LENGTH;
					new_base_addr = (block_idx + xgridlength*ygridlength)*blocksize;
				}
				_sol[idx.x][idx.y][tz+2] = sol_in[new_base_addr + new_tIdx];
			}

			__syncthreads();


			float a, b, c, oldT, newT;

			// compute new value
			oldT = newT = _sol[idx.x][idx.y][idx.z];

			if(isValid)
			{
				a = min(_sol[tx][idx.y][idx.z],_sol[tx+2][idx.y][idx.z]);
				b = min(_sol[idx.x][ty][idx.z],_sol[idx.x][ty+2][idx.z]);
				c = min(_sol[idx.x][idx.y][tz],_sol[idx.x][idx.y][tz+2]);

				float tmp = (float) get_time_eikonal(a, b, c, dh, F);
				newT = min(tmp,oldT);

				sol_out[base_addr + tIdx] = newT;
			}

			// write back to global memory
			float residue = oldT - newT;
			con[base_addr + tIdx] = (residue < EPS) ? true : false;	
		}
	}
}
